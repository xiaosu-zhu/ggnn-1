/* Copyright 2019 ComputerGraphics Tuebingen. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/
// Authors: Fabian Groh, Lukas Ruppert, Patrick Wieschollek, Hendrik P.A. Lensch
//
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <gflags/gflags.h>
#include <glog/logging.h>
#include <stdio.h>

#include <hipcub/hipcub.hpp>
#include <iostream>
#include <vector>

#include "ggnn/cuda_knn_ggnn.cuh"
#include "ggnn/utils/cuda_knn_constants.cuh"
#include "ggnn/utils/cuda_knn_dataset.cuh"
#include "ggnn/utils/cuda_knn_utils.cuh"

DEFINE_string(base_filename, "", "path to file with base vectors");
DEFINE_string(query_filename, "", "path to file with perform_query vectors");
DEFINE_string(groundtruth_filename, "",
              "path to file with groundtruth vectors");
DEFINE_string(graph_filename, "",
              "path to file that contains the serialized graph");
DEFINE_double(tau, 0.5, "Parameter tau");
DEFINE_int32(refinement_iterations, 2, "Number of refinement iterations");
DEFINE_int32(gpu_id, 0, "GPU id");
DEFINE_bool(grid_search, false,
            "Perform queries for a wide range of parameters.");

int main(int argc, char* argv[]) {
  google::InitGoogleLogging(argv[0]);
  google::LogToStderr();

  gflags::SetUsageMessage(
      "GGNN: Graph-based GPU Nearest Neighbor Search\n"
      "by Fabian Groh, Lukas Ruppert, Patrick Wieschollek, Hendrik P.A. "
      "Lensch\n"
      "(c) 2020 Computer Graphics University of Tuebingen");
  gflags::SetVersionString("1.0.0");
  google::ParseCommandLineFlags(&argc, &argv, true);

  CHECK(file_exists(FLAGS_base_filename))
      << "File for base vectors has to exist";
  CHECK(file_exists(FLAGS_query_filename))
      << "File for perform_query vectors has to exist";

  CHECK_GE(FLAGS_tau, 0) << "Tau has to be bigger or equal 0.";
  CHECK_GE(FLAGS_refinement_iterations, 0)
      << "The number of refinement iterations has to be non-negative.";

  // ####################################################################
  // compile-time configuration
  //
  // data types
  //
  /// data type for addressing points (needs to be able to represent N)
  using KeyT = int32_t;
  /// data type of the dataset (e.g., char, int, float)
  using BaseT = float;
  /// data type of computed distances
  using ValueT = float;
  /// data type for addressing base-vectors (needs to be able to represent N*D)
  using BAddrT = uint32_t;
  /// data type for addressing the graph (needs to be able to represent
  /// N*KBuild)
  using GAddrT = uint32_t;
  //
  // dataset configuration (here: SIFT1M)
  //
  /// dimension of the dataset
  const int D = 512;
  /// distance measure (Euclidean or Cosine)
  const DistanceMeasure measure = Cosine;
  //
  // search-graph configuration
  //
  /// number of neighbors per point in the graph
  const int KBuild = 24;
  /// maximum number of inverse/symmetric links (KBuild / 2 usually works best)
  const int KF = KBuild / 2;
  /// segment/batch size (needs to be > KBuild-KF)
  const int S = 32;
  /// graph height / number of layers (4 usually performs best)
  const int L = 4;
  //
  // query configuration
  //
  /// number of neighbors to search for
  const int KQuery = 10;

  static_assert(KBuild - KF < S,
                "there are not enough points to fill the local neighbor list!");

  LOG(INFO) << "Using the following parameters " << KBuild << " (KBuild) " << KF
            << " (KF) " << S << " (S) " << L << " (L) " << D << " (D) ";

  // Set the requested GPU id, if possible.
  {
    int numGpus;
    hipGetDeviceCount(&numGpus);
    CHECK_GE(FLAGS_gpu_id, 0) << "This GPU does not exist";
    CHECK_LT(FLAGS_gpu_id, numGpus) << "This GPU does not exist";

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, FLAGS_gpu_id);
    LOG(INFO) << "device name: " << prop.name;
  }
  hipSetDevice(FLAGS_gpu_id);

  typedef GGNN<measure, KeyT, ValueT, GAddrT, BaseT, BAddrT, D, KBuild, KF,
               KQuery, S>
      GGNN;
  GGNN m_ggnn{FLAGS_base_filename, FLAGS_query_filename,
              FLAGS_groundtruth_filename, L, static_cast<float>(FLAGS_tau)};

  m_ggnn.ggnnMain(FLAGS_graph_filename, FLAGS_refinement_iterations);

  auto query_function = [&m_ggnn](const float tau_query) {
    hipMemcpyToSymbol(HIP_SYMBOL(c_tau_query), &tau_query, sizeof(float));
    LOG(INFO) << "--";
    LOG(INFO) << "Query with tau_query " << tau_query;
    // faster for C@1 = 99%
    LOG(INFO) << "fast query (good for C@1)";
    m_ggnn.queryLayer<32, 200, 256, 64>();
    // better for C@10 > 99%
    LOG(INFO) << "regular query (good for C@10)";
    m_ggnn.queryLayer<32, 400, 448, 64>();
    // expensive, can get to 99.99% C@10
    // m_ggnn.queryLayer<128, 2000, 2048, 256>();
  };

  if (FLAGS_grid_search) {
    LOG(INFO) << "--";
    LOG(INFO) << "grid-search:";
    for (int i = 0; i < 70; ++i) query_function(i * 0.01f);
    for (int i = 7; i <= 20; ++i) query_function(i * 0.1f);
  } else {  // by default, just execute a few queries
    LOG(INFO) << "--";
    LOG(INFO) << "90, 95, 99% R@1, 99% C@10 (using -tau 0.5 "
                 "-refinement_iterations 2):";
    query_function(0.34f);
    query_function(0.41f);
    query_function(0.51f);
    query_function(0.64f);
  }

  printf("done! \n");
  gflags::ShutDownCommandLineFlags();
  return 0;
}
