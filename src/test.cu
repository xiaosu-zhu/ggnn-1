/* Copyright 2019 ComputerGraphics Tuebingen. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/
// Authors: Fabian Groh, Lukas Ruppert, Patrick Wieschollek, Hendrik P.A. Lensch
//

#ifndef CUDA_API_PER_THREAD_DEFAULT_STREAM
#define CUDA_API_PER_THREAD_DEFAULT_STREAM
#endif

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <gflags/gflags.h>
#include <glog/logging.h>
#include <stdio.h>

#include <hipcub/hipcub.hpp>

// only needed for file_exists check
#include <sys/stat.h>

inline bool file_exists(const std::string& name) {
  struct stat buffer;
  return (stat(name.c_str(), &buffer) == 0);
}

#include <iostream>
#include <vector>

#include "ggnn/cuda_knn_ggnn_multi_gpu.cuh"
#include "ggnn/utils/cuda_knn_constants.cuh"

DEFINE_string(
    mode, "bq",
    "Mode: bq -> build_and_query, bs -> build_and_store, lq -> load_and_query");
DEFINE_string(base_filename, "", "path to file with base vectors");
DEFINE_string(query_filename, "", "path to file with perform_query vectors");
DEFINE_string(groundtruth_filename, "",
              "path to file with groundtruth vectors");
DEFINE_string(graph_dir, "./", "directory to store and load ggnn graph files.");
DEFINE_double(tau, 0.5, "Parameter tau");
DEFINE_int32(factor, 231104, "Factor");
DEFINE_int32(base, 1, "N_base: base x factor");
DEFINE_int32(shard, 1, "N_shard: shard x factor");
DEFINE_int32(refinement_iterations, 2, "Number of refinement iterations");
DEFINE_string(gpu_ids, "0", "GPU id");
DEFINE_bool(grid_search, false,
            "Perform queries for a wide range of parameters.");

int main(int argc, char* argv[]) {
  google::InitGoogleLogging(argv[0]);
  google::LogToStderr();

  gflags::SetUsageMessage(
      "GGNN: Graph-based GPU Nearest Neighbor Search\n"
      "by Fabian Groh, Lukas Ruppert, Patrick Wieschollek, Hendrik P.A. "
      "Lensch\n"
      "(c) 2020 Computer Graphics University of Tuebingen");
  gflags::SetVersionString("1.0.0");
  google::ParseCommandLineFlags(&argc, &argv, true);

  LOG(INFO) << "Reading files";
  CHECK(file_exists(FLAGS_base_filename))
      << "File for base vectors has to exist";
  CHECK(file_exists(FLAGS_query_filename))
      << "File for perform_query vectors has to exist";
  CHECK(file_exists(FLAGS_groundtruth_filename))
      << "File for groundtruth vectors has to exist";

  CHECK_GE(FLAGS_tau, 0) << "Tau has to be bigger or equal 0.";
  CHECK_GE(FLAGS_refinement_iterations, 0)
      << "The number of refinement iterations has to be non-negative.";

  // ####################################################################
  // compile-time configuration
  //
  // data types
  //
  /// data type for addressing points (needs to be able to represent N)
  using KeyT = int32_t;
  /// data type of the dataset (e.g., char, int, float)
  using BaseT = float;
  /// data type of computed distances
  using ValueT = float;
  /// data type for addressing base-vectors (needs to be able to represent N*D)
  using BAddrT = uint32_t;
  /// data type for addressing the graph (needs to be able to represent
  /// N*KBuild)
  using GAddrT = uint32_t;
  //
  // dataset configuration (here: SIFT1M)
  //
  /// dimension of the dataset
  const int D = 384;
  /// distance measure (Euclidean or Cosine)
  const DistanceMeasure measure = Cosine;
  //
  // search-graph configuration
  //
  /// number of neighbors per point in the graph
  const int KBuild = 24;
  /// maximum number of inverse/symmetric links (KBuild / 2 usually works best)
  const int KF = KBuild / 2;
  /// segment/batch size (needs to be > KBuild-KF)
  const int S = 32;
  /// graph height / number of layers (4 usually performs best)
  const int L = 4;
  //
  // query configuration
  //
  /// number of neighbors to search for
  const int KQuery = 1;

  static_assert(KBuild - KF < S,
                "there are not enough points to fill the local neighbor list!");

  LOG(INFO) << "Using the following parameters " << KBuild << " (KBuild) " << KF
            << " (KF) " << S << " (S) " << L << " (L) " << D << " (D) ";

  std::istringstream iss(FLAGS_gpu_ids);
  std::vector<std::string> results(std::istream_iterator<std::string>{iss},
                                   std::istream_iterator<std::string>());

  int numGpus;
  hipGetDeviceCount(&numGpus);

  std::vector<int> gpus;
  for (auto&& r : results) {
    int gpu_id = atoi(r.c_str());
    printf("GPU %d: ", gpu_id);
    {
      CHECK_GE(gpu_id, 0) << "This GPU does not exist";
      CHECK_LT(gpu_id, numGpus) << "This GPU does not exist";

      hipDeviceProp_t prop;
      hipGetDeviceProperties(&prop, gpu_id);
      printf("Found device name: %s\n", prop.name);

      gpus.push_back(gpu_id);
    }
  }

  const size_t N_base = FLAGS_base * FLAGS_factor;
  const int N_shard = FLAGS_shard * FLAGS_factor;

  typedef GGNNMultiGPU<measure, KeyT, ValueT, GAddrT, BaseT, BAddrT, D, KBuild,
                       KF, KQuery, S>
      GGNN;
  GGNN ggnn{
      FLAGS_base_filename,
      FLAGS_query_filename,
      file_exists(FLAGS_groundtruth_filename) ? FLAGS_groundtruth_filename : "",
      L,
      static_cast<float>(FLAGS_tau),
      N_base};

  ggnn.ggnnMain(gpus, FLAGS_mode, N_shard, FLAGS_graph_dir,
                FLAGS_refinement_iterations, FLAGS_grid_search);

  printf("done! \n");
  gflags::ShutDownCommandLineFlags();
  return 0;
}
